#include "hip/hip_runtime.h"
#include <stdio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#define TIMING_SUPPORT

#ifdef TIMING_SUPPORT
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#endif

#define CUDA_TIMING

/**
    CUDA Kernel Device code
    -simple parallel code
    -no optimisation
*/
__global__ void
p_vectorAverageSimple(const float *A, float *B, int numElements, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int counter = 0;
    B[i] = 0;

    if (i < numElements) {
    	for(int j = 0; j < n; j++) {
    		int index = (i-n+1+j);
            if(index >= 0) {
    		    B[i] += A[max(0,index)];
                counter++;
            }
    	}
        B[i] = B[i]/(float)counter;
    }
}

#define BLOCK_SIZE 1024

__global__ void
p_scan(float *X, float *Y, int len) {
	__shared__ float XY[BLOCK_SIZE*2]; // 2 buffers
	int rBuf = 0, wBuf = BLOCK_SIZE;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < len) {
		XY[wBuf + threadIdx.x] = X[i];
	}

	for(uint s = 1; s < BLOCK_SIZE; s*= 2) {
		__syncthreads();
		wBuf = BLOCK_SIZE - wBuf;
		rBuf = BLOCK_SIZE - rBuf;
		if(threadIdx.x >= s) {
			XY[wBuf+threadIdx.x] = XY[rBuf+threadIdx.x - s] + XY[rBuf+threadIdx.x];
		}
		else {
			// If not adding, thread should copy
			XY[wBuf+threadIdx.x] = XY[rBuf+threadIdx.x];
		}
	}
	if(i < len) {
		Y[i] = XY[wBuf + threadIdx.x];
	}
}

__global__ void
p_vectorAverageInefficient(const float *A, float *B, int numElements, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i >= n) {
		int index = i-n;
		B[i] = A[i]-A[index];
		B[i] = B[i]/n;
	}
	else {
		B[i] = A[i]/(float)(i+1);
	}
}



/**
    Device code for average
*/
void
s_vectorAverageSimple(const float *A, float *B, int numElements, int n) {
    for(int i = 0; i < numElements; i++) {
    	int counter = 0;
        B[i] = 0;
    	for(int j = 0; j < n; j++) {
    		int index = (i-n+1+j);
    		if(index >= 0) {
                B[i] += A[max(0,index)];
    			counter++;
    		}
    	}
    	B[i] = B[i]/(float)counter;
    }
}

/**
	Device code for scan
 */
void
s_scan(const float *x, float *y, int len) {
	y[0] = x[0];
	for(int i = 0; i < len; i++) {
		y[i] = y[i-1] + x[i];
	}
}

void
s_vectorAverageEfficient(const float *A, float *B, float *C, int numElements, int n) {
	s_scan(A, C, numElements);
	for(int i = 0; i < numElements; i++) {
		if(i >= n) {
			int index = i-n;
			B[i] = C[i]-C[index];
			B[i] = B[i]/n;
		}
		else {
			B[i] = C[i]/(float)(i+1);
		}
	}
}



/**
    Host main routine
*/
int
main(void) {
    // Error code check for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used and compute size
    int numElements = 10;
    int windowSize = 5;
    size_t size = numElements * sizeof(float);
    printf("[Vector average of %d elements]\n", numElements);

    // Allocate host input vector A
    float *h_A = (float *)malloc(size);
    // Allocate host input vector B
    float *h_B = (float *)malloc(size);
    // Allocate host input vector C - used for serial version to check parallel
    float *h_C = (float *)malloc(size);
    // Allocate host input vector D - used for serial version to check parallel
    float *h_D = (float *)malloc(size);

    // Verify success of allocations
    if(h_A == NULL || h_B == NULL || h_C == NULL || h_D == NULL) {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    } 

    // Initialise host vectors
    for(int i = 0; i < numElements; i++) {
        h_A[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device inp ut vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

    // Copy the host input vector A in host memory to the device input vectors in
	// device memory
	printf("Copy input data from the host memory to the CUDA device\n");
	err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	// Launch the Vector Average CUDA Kernel
	int nIter = 1; // Number of iterations to run the kernel
	int threadsPerBlock = 1024;
    // Note this pattern, based on integer division, for rounding up
    int blocksPerGrid = 1 + ((numElements - 1) / threadsPerBlock);

    printf("%d iterated launches of the CUDA kernel with %d blocks of %d threads\n",
        		nIter, blocksPerGrid, threadsPerBlock);

#ifdef TIMING_SUPPORT
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);             // create a timer
    sdkStartTimer(&timer);               // start the timer
#endif
#ifdef CUDA_TIMING
    hipEvent_t start, stop;
    float time;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord( start, 0 );
#endif

    // Call serial version
    //s_vectorAverageSimple(h_A, h_C, numElements, windowSize);
    s_vectorAverageEfficient(h_A, h_C, h_D, numElements, windowSize);
    //s_scan(h_A, h_C, numElements);

    //for (int j = 0; j < nIter; j++)

        // Call kernel version
        //p_vectorAverageSimple<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, numElements, windowSize);
    	p_scan<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, numElements);
    	p_vectorAverageInefficient<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, numElements, windowSize);

#ifdef CUDA_TIMING
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );

    err = hipEventElapsedTime( &time, start, stop );
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get elapsed time (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipEventDestroy( start );
    hipEventDestroy( stop );
    printf("CUDA_TIMING: %.4f ms\n", time);
#endif

    // wait for device to finish
    hipDeviceSynchronize();
    err = hipGetLastError();

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


#ifdef TIMING_SUPPORT
    // stop and destroy timer
    sdkStopTimer(&timer);
    double dSeconds = sdkGetTimerValue(&timer)/(1000.0);
    double dNumOps = 1.0e-9 * nIter * size;
    double gflops = dNumOps/dSeconds;

    //Log throughput, etc

    printf("Throughput = %.4f GFlops\nTime = %.5f s\nSize = %.5f Gops\n\n",
  	   gflops, dSeconds, dNumOps);
    sdkDeleteTimer(&timer);
#endif


// Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector B from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Print result
    for(int i = 0; i < numElements; i++) {
        printf("%lf\n", h_C[i]);
    }

	printf("\n");

    for(int i = 0; i < numElements; i++) {
        printf("%lf\n", h_B[i]);
    }


    /*
    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i) {
        // fabs function gets absolute value
        if (fabs(h_B[i] - h_C[i]) > 1e-2) {
        	printf("%lf\n%lf", h_B[i], h_C[i]);
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Test PASSED\n");
    */



    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_B);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_D);

    // Reset the device and exit
    err = hipDeviceReset();

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to reset the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    printf("Done\n");
    return 0;
}
