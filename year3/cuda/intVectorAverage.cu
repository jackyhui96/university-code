#include "hip/hip_runtime.h"
/*
    - Jacky Hui, 14435743
    - Goals Achieved:
        [x] scan for small arrays
        [x] scan for large arrays
        [x] windowed average calculation from scaned output vector
    - Initial speed up: 6.97
    - Improved speed up: 9.38
        - Tried different configurations of block sizes, initially was 1024, found that 256 is best
*/

#include <stdio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#define TIMING_SUPPORT

#ifdef TIMING_SUPPORT
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#endif

#define CUDA_TIMING

/**
    CUDA Kernel Device code
*/
#define BLOCK_SIZE 256

__global__ void
p_scanEfficient(int *X, int *Y, int len) {
	__shared__ int XY[BLOCK_SIZE];

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < len) {
		XY[threadIdx.x] = X[i];
	}
	// Reduction phase
	for(uint stride = 1; stride < blockDim.x; stride *= 2)
	{
		__syncthreads();
		uint index = (threadIdx.x + 1) * stride * 2 - 1;
		if(index < blockDim.x) 
			XY[index] += XY[index - stride];
	}

	// Distribution phase
	for(uint stride = BLOCK_SIZE/4; stride > 0; stride /= 2)
	{
		__syncthreads();
		uint index = (threadIdx.x + 1) * stride * 2 - 1;
		if(index + stride < BLOCK_SIZE)
			XY[index + stride] += XY[index];
	}

	__syncthreads();

	if(i < len)
		Y[i] = XY[threadIdx.x];
}

__global__ void
p_vectorAverage(const int *A, float *B, int numElements, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i >= n) {
		int index = i-n+1;
		B[i] = (float)(A[i]-A[index])/(float)n;
	}
	else {
		B[i] = (float)A[i]/(float)(i+1);
	}
}

__global__ void
p_extract(const int *A, int *B) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if((i+1) % BLOCK_SIZE == 0) {
        int index = ((i+1) / BLOCK_SIZE) - 1;
        B[index] = A[i];
    }
}

__global__ void
p_addToBlocks(const int *A, int *B) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i >= BLOCK_SIZE) {
        int index = (i / BLOCK_SIZE) - 1;
        B[i] = B[i] + A[index];
    }
}

/**
	Serial code for scan
*/
void
s_scan(const int *x, int *y, int len) {
	y[0] = x[0];
	for(int i = 0; i < len; i++) {
		y[i] = y[i-1] + x[i];
	}
}

/**
    Serial code for average
*/
void
s_vectorAverageEfficient(const int *A, float *B, int *C, int numElements, int n) {
	s_scan(A, C, numElements);
	for(int i = 0; i < numElements; i++) {
		if(i >= n) {
			int index = i-n+1;
			B[i] = C[i]-C[index];
			B[i] = B[i]/n;
		}
		else {
			B[i] = C[i]/(float)(i+1);
		}
	}
}

/**
    Host main routine
*/
int
main(void) {
    // Error code check for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used and compute size
    int numElements = 10000000;
    int windowSize = 5;
    size_t size = numElements * sizeof(int);
    size_t sizeF = numElements * sizeof(float);
    printf("[Vector average of %d elements]\n", numElements);

    // Allocate host input vector A
    int *h_A = (int *)malloc(size);
    // Allocate host input vector B
    float *h_B = (float *)malloc(sizeF);
    // Allocate host input vector C - used for serial version to check parallel
    float *h_C = (float *)malloc(sizeF);
    // Allocate host input vector D - used for serial version to check parallel
    int *h_D = (int *)malloc(size);
    // Allocate host input vector E - used for extract
    int *h_E = (int *)malloc(size);

    // Verify success of allocations
    if(h_A == NULL || h_B == NULL || h_C == NULL || h_D == NULL || h_E == NULL) {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    } 

    // Initialise host vectors - random ints between 0-9
    for(int i = 0; i < numElements; i++) {
        h_A[i] = rand()%10;
    }

    // Allocate the device input vector A
    int *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
    // Allocate the device input vector B
    int *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
    // Allocate the device input vector C
	int *d_C = NULL;
	err = hipMalloc((void **)&d_C, size);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

    // Allocate the device input vector E
	int *d_E = NULL;
	err = hipMalloc((void **)&d_E, size);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector E (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

    // Allocate the device input vector E2 - scanned result of extract
	int *d_E2 = NULL;
	err = hipMalloc((void **)&d_E2, size);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector E2 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

    // Allocate the device input vector E2 - scanned result of extract
	int *d_E3 = NULL;
	err = hipMalloc((void **)&d_E3, size);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector E3 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

    // Allocate the device input vector E2 - scanned result of extract
	int *d_E4 = NULL;
	err = hipMalloc((void **)&d_E4, size);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector E4 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

    // Allocate the device input vector B2 - windowed average of b
	float *d_B2 = NULL;
	err = hipMalloc((void **)&d_B2, sizeF);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate device vector B2 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

    // Copy the host input vector A in host memory to the device input vectors in
	// device memory
	printf("Copy input data from the host memory to the CUDA device\n");
	err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}


	// Launch the Vector Average CUDA Kernel
	int threadsPerBlock = BLOCK_SIZE;
    // Note this pattern, based on integer division, for rounding up
    int blocksPerGrid = 1 + ((numElements - 1) / threadsPerBlock);

    printf("Launch  CUDA kernel with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

#ifdef TIMING_SUPPORT
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);             // create a timer
    sdkStartTimer(&timer);              // start the timer
#endif
#ifdef CUDA_TIMING
    hipEvent_t start, stop;
    float time;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord( start, 0 );
#endif

    // Call serial version
    //s_vectorAverageEfficient(h_A, h_C, h_D, numElements, windowSize);
    //s_scan(h_A, h_C, numElements);

    // Call kernel version
    p_scanEfficient<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, numElements);
    p_extract<<<blocksPerGrid, threadsPerBlock>>>(d_B, d_E);
    p_scanEfficient<<<blocksPerGrid, threadsPerBlock>>>(d_E, d_E2, (numElements/BLOCK_SIZE));

    if(numElements > BLOCK_SIZE*BLOCK_SIZE) {
        p_extract<<<blocksPerGrid, threadsPerBlock>>>(d_E2, d_E3);
        p_scanEfficient<<<blocksPerGrid, threadsPerBlock>>>(d_E3, d_E4, (numElements/BLOCK_SIZE/BLOCK_SIZE));
        p_addToBlocks<<<blocksPerGrid, threadsPerBlock>>>(d_E4, d_E2);
    }
    
    p_addToBlocks<<<blocksPerGrid, threadsPerBlock>>>(d_E2, d_B);
    p_vectorAverage<<<blocksPerGrid, threadsPerBlock>>>(d_B, d_B2, numElements, windowSize);

    hipDeviceSynchronize();   // Make sure kernels have finished before stopping timer

#ifdef CUDA_TIMING
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );

    err = hipEventElapsedTime( &time, start, stop );
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to get elapsed time (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipEventDestroy( start );
    hipEventDestroy( stop );
    printf("CUDA_TIMING: %.4f ms\n", time);
#endif

    // wait for device to finish
    hipDeviceSynchronize();
    err = hipGetLastError();

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

#ifdef TIMING_SUPPORT
    // stop and destroy timer
    sdkStopTimer(&timer);
    double dSeconds = sdkGetTimerValue(&timer)/(1000.0);
    double dNumOps = 1.0e-9 * size;
    double gflops = dNumOps/dSeconds;

    //Log throughput, etc

    printf("Throughput = %.4f GFlops\nTime = %.5f s\nSize = %.5f Gops\n\n",
  	   gflops, dSeconds, dNumOps);
    sdkDeleteTimer(&timer);
#endif

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_B, d_B2, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector B from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_E, d_E2, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector E2 from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Run serial version to check against parallel
    s_vectorAverageEfficient(h_A, h_C, h_D, numElements, windowSize);
    //s_scan(h_A, h_C, numElements);

/*
    // Print result - Debugging
    printf("Serial Version - Vector Average\n");
    for(int i = 0; i < numElements; i++) {
        printf("%d\n", h_C[i]);
    }
	printf("\n");

	printf("Parallel Version - Vector Average\n");
    for(int i = 0; i < numElements; i++) {
        printf("%lf\n", h_B[i]);
    }
    printf("\n");

	printf("Parallel Version - Extract Result\n");
    for(int i = 0; i < (numElements/BLOCK_SIZE); i++) {
        printf("%d\n", h_E[i]);
    }
*/

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i) {
        // fabs function gets absolute value
        if (fabs(h_B[i] - h_C[i]) > 1e-2) {
        	printf("%lf\n%lf", h_B[i], h_C[i]);
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Test PASSED\n");


    // Free device global memory
    err = hipFree(d_A);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);
    if (err != hipSuccess) {
		fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
    }

    err = hipFree(d_E);
    if (err != hipSuccess) {
		fprintf(stderr, "Failed to free device vector E (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
    }

    err = hipFree(d_E2);
    if (err != hipSuccess) {
		fprintf(stderr, "Failed to free device vector E2 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
    }

    err = hipFree(d_E3);
    if (err != hipSuccess) {
		fprintf(stderr, "Failed to free device vector E3 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
    }

    err = hipFree(d_E4);
    if (err != hipSuccess) {
		fprintf(stderr, "Failed to free device vector E4 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
    }

    err = hipFree(d_B2);
    if (err != hipSuccess) {
		fprintf(stderr, "Failed to free device vector B2 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_D);
    free(h_E);

    // Reset the device and exit
    err = hipDeviceReset();

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to reset the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    printf("Done\n");
    return 0;
}
